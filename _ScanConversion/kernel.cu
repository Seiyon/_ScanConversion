#include "hip/hip_runtime.h"
﻿
#include "stdafx.h"
#include "kernel.cuh"


__global__ void ScanConversionKernel(
    const char* const input, 
    char* const output
    )
{
    int i = threadIdx.x;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t ScanConversion(const char* const input, dataInfo* inputInfo, char* const output, dataInfo* outputInfo)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        throw std::logic_error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    //define auto memory free pointer
    std::shared_ptr<char> gInput(AllocCudaMem(inputInfo), hipFree);
    std::shared_ptr<char> gOutput(AllocCudaMem(outputInfo), hipFree);

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(gInput.get(), input, GetTotalSize(inputInfo), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        throw std::logic_error("hipMemcpy failed!");
    }

    dim3 block(512,1,1); //thread x, y, z count
    dim3 grid(inputInfo->width / block.x , inputInfo->heigth / block.y , 1 / block.z); // block x, y, z count
    //function

    ScanConversionKernel <<< block, grid >>> (gInput.get(), gOutput.get());

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::stringstream stm;
        stm << "addKernel launch failed: " << hipGetErrorString(cudaStatus) << "\n";
        throw std::logic_error(stm.str());
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::stringstream stm;
        stm << "hipDeviceSynchronize returned error code " << (int)cudaStatus << " after launching addKernel!\n";
        throw std::logic_error(stm.str());
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(output, gOutput.get(), GetTotalSize(outputInfo), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        throw std::logic_error("hipMemcpy failed!");
    }

    return cudaStatus; // SUCCESS status return
}

char* AllocCudaMem(dataInfo* inputInfo)
{
    hipError_t cudaStatus;

    char* temp;
    cudaStatus = hipMalloc((void**)&temp, GetTotalSize(inputInfo));
    if (cudaStatus != hipSuccess) {
        hipFree(temp);
        throw std::logic_error("CUDA malloc error");
    }

    return temp;
}

int GetTotalSize(dataInfo* info)
{
    if(info == nullptr) throw std::logic_error("info is null");

    //data size calculation could be exceed data format(32bit signed). need to carefully use data type which is returned

    return info->heigth * info->unitDataSize * info->unitDataSize;
}
